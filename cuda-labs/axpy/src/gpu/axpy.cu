#include "hip/hip_runtime.h"
extern "C" {
#include "axpy.h"
}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

const int block_size = 256;

__global__
void saxpy_kernel(int n, float a, float *x, int incx, float *y, int incy) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i*incx < n && i*incy < n)
        y[i*incy] += x[i*incx]*a;
}

__global__
void daxpy_kernel(int n, double a, double *x, int incx, double *y, int incy) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i*incx < n && i*incy < n)
        y[i*incy] += x[i*incx]*a;
}

extern "C"
void saxpy_gpu(int n, float a, float *x, int incx, float *y, int incy) {
    float *x_gpu, *y_gpu;
    hipMalloc((void**)&x_gpu, n*sizeof(float));
    hipMalloc((void**)&y_gpu, n*sizeof(float));

    hipMemcpy(x_gpu, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y, n*sizeof(float), hipMemcpyHostToDevice);

    int num_blocks = (n + block_size - 1)/block_size;

    saxpy_kernel <<< num_blocks, block_size >>> (n,a,x_gpu,incx,y_gpu,incy);

    hipMemcpy(y, y_gpu, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);
}

extern "C"
void daxpy_gpu(int n, double a, double *x, int incx, double *y, int incy) {
    double *x_gpu, *y_gpu;
    hipMalloc((void**)&x_gpu, n*sizeof(double));
    hipMalloc((void**)&y_gpu, n*sizeof(double));

    hipMemcpy(x_gpu, x, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y, n*sizeof(double), hipMemcpyHostToDevice);

    int num_blocks = (n + block_size - 1)/block_size;

    daxpy_kernel <<< num_blocks, block_size >>> (n,a,x_gpu,incx,y_gpu,incy);

    hipMemcpy(y, y_gpu, n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);
}
