#include "hip/hip_runtime.h"
extern "C" {
#include "gemm.h"
}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

const int BLOCK_SIZE = 16;

__global__
void blas_sgemm_kernel(const int m, const int n, const int k, const float *a,
            const float *b, float *c) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int ai = BLOCK_SIZE*n*by + n*ty;
    int bj = BLOCK_SIZE*bx + tx;

    float sum = 0.0f;
    for (int ki = 0; ki < n; ki++) {
        sum += a[ai + ki]*b[ki*k + bj];
    }

    c[by*k*BLOCK_SIZE + ty*k + bx*BLOCK_SIZE + tx] = sum;
}

__global__
void blas_sgemm_optimized_kernel(const int m, const int n, const int k, const float *a,
            const float *b, float *c) {

    const int row = threadIdx.x;
    const int col = threadIdx.y;
    const int globalRow = BLOCK_SIZE*blockIdx.x + row;
    const int globalCol = BLOCK_SIZE*blockIdx.y + col;

    __shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];

    float acc = 0.0f;

    const int numTiles = k/BLOCK_SIZE;
    for (int t=0; t<numTiles; t++) {

        const int tiledRow = BLOCK_SIZE*t + row;
        const int tiledCol = BLOCK_SIZE*t + col;
        as[col][row] = a[globalRow*n + tiledCol];
        bs[col][row] = b[tiledRow*k + globalCol];

        __syncthreads();
        for (int i = 0; i < BLOCK_SIZE; i++) {
            acc += as[i][row]*bs[col][i];
        }
        __syncthreads();
    }

    c[globalRow*k + globalCol] = acc;
}


extern "C"
void cblas_sgemm_gpu_cuda(const int m, const int n, const int k, const float *a,
            const float *b, float *c, const mode_opt opt) {
    float *a_gpu, *b_gpu, *c_gpu;
    hipMalloc((void**)&a_gpu, m*n*sizeof(float));
    hipMalloc((void**)&b_gpu, n*k*sizeof(float));
    hipMalloc((void**)&c_gpu, m*k*sizeof(float));

    hipMemcpy(a_gpu, a, m*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, n*k*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c, m*k*sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(m / dimBlock.x, k / dimBlock.y);
    if (opt == OPTIMIZED)
        blas_sgemm_optimized_kernel <<< dimGrid, dimBlock >>> (m, n, k, a_gpu, b_gpu, c_gpu);
    else
        blas_sgemm_kernel <<< dimGrid, dimBlock >>> (m, n, k, a_gpu, b_gpu, c_gpu);

    hipMemcpy(c, c_gpu, m*k*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
}
