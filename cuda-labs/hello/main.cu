
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void kernel()
{
  printf("Hello, world!\n");
}

int main()
{
  kernel<<<2,2>>>();
  hipDeviceSynchronize();
  return 0;
}